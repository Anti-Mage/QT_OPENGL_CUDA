#include <stdio.h>
#include <hip/hip_runtime.h>



#define N 100

__global__ void kernel(int *a, int *b, int *c)
{
	int globalID = threadIdx.x + blockIdx.x * blockDim.x;
	printf("globalID = %d\n", globalID);
	if (globalID < N)
		c[globalID] = a[globalID] + b[globalID];

}

extern "C" void launch_kernel()
{
 printf("RUN CUDA KERNEL\n");


int a[N], b[N], c[N];
int *dev_a, *dev_b, *dev_c;
hipMalloc((void **) &dev_a, N*sizeof(int));
hipMalloc((void **) &dev_b, N*sizeof(int));
hipMalloc((void **) &dev_c, N*sizeof(int));
// Fill Arrays
for (int i = 0; i < N; i++)
{
a[i] = i,
b[i] = 1;
}
hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
kernel<<<N,1>>>(dev_a, dev_b, dev_c);
hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
for (int i = 0; i < N; i++)
{
printf("%d + %d = %d\n", a[i], b[i], c[i]);
}

}